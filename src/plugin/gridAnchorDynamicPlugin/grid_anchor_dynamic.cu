#include "hip/hip_runtime.h"
#include "amir_cuda_util/cuda_util.h"
#include "grid_anchor_dynamic.h"
#include <algorithm>
#include <cmath>
#include <stdio.h>


namespace amirstan {
namespace plugin {
using namespace amirstan::cuda;
template <typename T>
__global__ void grid_anchor_dynamic_kernel(T *output, const T *base_anchor,
                                           int width, int height, int stride,
                                           int num_base_anchor) {
  CUDA_KERNEL_LOOP(i, width * height * num_base_anchor) {
    const int y = i / (width * num_base_anchor);
    const int x = (i % (width * num_base_anchor)) / num_base_anchor;
    const int base_id = i % num_base_anchor;

    output[i * 4 + 0] = base_anchor[base_id * 4 + 0] + x * stride;
    output[i * 4 + 1] = base_anchor[base_id * 4 + 1] + y * stride;
    output[i * 4 + 2] = base_anchor[base_id * 4 + 2] + x * stride;
    output[i * 4 + 3] = base_anchor[base_id * 4 + 3] + y * stride;
  }
}

template <typename T>
void grid_anchor_dynamic(T *output, const T *base_anchor, int width, int height,
                         int stride, int num_base_anchor, hipStream_t stream) {

  size_t input_size = num_base_anchor * height * width;
  grid_anchor_dynamic_kernel<T>
      <<<GET_BLOCKS(input_size), CUDA_NUM_THREADS, 0, stream>>>(
          output, base_anchor, width, height, stride, num_base_anchor);
}

template void grid_anchor_dynamic<float>(float *output,
                                         const float *base_anchor, int width,
                                         int height, int stride,
                                         int num_base_anchor,
                                         hipStream_t stream);

} // namespace plugin
} // namespace amirstan
